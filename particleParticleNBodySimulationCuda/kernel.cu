#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "Particle.h"
#include "Vector.h"
#include <fstream>
#include <filesystem>

/*Решение задачи N - тел методом Particle - Particle(пр¤мого интегрировани¤).
моделируетс¤ трёхмерное пространство,
обезразмеривание осуществлено с соображением G = 1,
уравнения движения решаются методом Эйлера.
используется симметрично-противоположна¤ матрица гравитационных взаимодействий force,
благодар¤ которой количество необходимых вычислений уменьшаетс¤ в два раза,
в соответствии с третьим законом Ќьютона (Fij = -Fji)*/

/*Solution of N-body problem with Particle-Particle (direct sum) method.
This program calculates in 3D space,
with nondimensialization (G = 1),
equations of motion are solved with Euler's method.
Forces Fij and Fji are treated as equal with different signs, due to Newton's 3rd law. That halves the amount of calculations.*/


Particle* InitializeNBodySystem(const std::string path, int& n);

double Cube(double number);

Vector Sum(Vector* sequence, int size);
Vector Sum(Vector* sequence, int first, int size);

__global__ void calculateForce(Vector* force, Particle* particles, const size_t size)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < size && col < size && row < col)
	{
		double distanceX = particles[col].position.x - particles[row].position.x;
		double distanceY = particles[col].position.y - particles[row].position.y;
		double distanceZ = particles[col].position.z - particles[row].position.z;

		double vector = sqrt(distanceX * distanceX + distanceY * distanceY + distanceZ * distanceZ);
		double denominator = vector * vector * vector;

		force[row * size + col].x = distanceX * particles[row].mass * particles[col].mass / denominator;
		force[row * size + col].y = distanceY * particles[row].mass * particles[col].mass / denominator;
		force[row * size + col].z = distanceZ * particles[row].mass * particles[col].mass / denominator;

		force[col * size + row].x = -force[row * size + col].x;
		force[col * size + row].y = -force[row * size + col].y;
		force[col * size + row].z = -force[row * size + col].z;
	}
}

int main()
{
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return 1;
	}

	int n;
	double timeStep = 0.01;

	Particle* particles = InitializeNBodySystem("Particles.txt", n);

	Vector* force = new Vector[n * n];

	const size_t sizeForceBytes = n * n * sizeof(Vector);
	const size_t sizeParticlesBytes = n * sizeof(Particle);

	Particle* particlesDevice;
	Vector* forceDevice;

	const int blockSize = 32;

	dim3 dimBlock(blockSize, blockSize);
	dim3 dimGrid(n / blockSize + 1, n / blockSize + 1);
	

	std::filesystem::path path = L"coordinates";
	if (std::filesystem::exists(path))
	{
		std::filesystem::remove_all(path);
	}

	if (!std::filesystem::create_directory(path))
	{
		printf("Error making a directory\n");
		return 1;
	}

	double time = 0.0;
	for (;;)
	{
		std::ofstream fileCoordinates;
		std::string timeStr = std::to_string(time);
		fileCoordinates.open("coordinates\\" + timeStr + ".csv");
		fileCoordinates << "x;y;z\n";

		cudaStatus = hipMalloc((void**)&particlesDevice, sizeParticlesBytes);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			return 1;
		}

		cudaStatus = hipMalloc((void**)&forceDevice, sizeForceBytes);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			return 1;
		}

		cudaStatus = hipMemcpy(particlesDevice, particles, sizeParticlesBytes, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}


		for (int i = 0; i < n; ++i)
		{
			force[i * n + i].SetZeroVector();
		}

		calculateForce <<<dimGrid, dimBlock>>> (forceDevice, particlesDevice, n);
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			return -1;
		}

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			return 1;
		}

		cudaStatus = hipMemcpy(force, forceDevice, sizeForceBytes, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			return 1;
		}



		for (int i = 0; i < n; ++i)
		{
			fileCoordinates << particles[i].position << "\n";

			particles[i].acceleration = Sum(force, i * n, i * n + n) / particles[i].mass;

			particles[i].velocity = particles[i].velocity + particles[i].acceleration * timeStep;

			particles[i].position = particles[i].position + particles[i].velocity * timeStep;
		}

		time += timeStep;

		hipFree(particlesDevice);
		hipFree(forceDevice);

		fileCoordinates.close();
	}


	delete[] force;
	delete[] particles;
	return 0;
}

Particle* InitializeNBodySystem(const std::string path, int& n)
{
	std::ifstream fileParticles;
	fileParticles.open(path);

	char tempString[256];
	fileParticles.getline(tempString, 256, ':');

	fileParticles >> n;
	Particle* particles = new Particle[n];

	fileParticles.get();
	fileParticles.get();

	fileParticles.getline(tempString, 256);

	for (int i = 0; i < n; ++i)
	{
		fileParticles >> particles[i].mass;
		fileParticles.get();
		fileParticles >> particles[i].velocity.x >> particles[i].velocity.y >> particles[i].velocity.z;
		fileParticles.get();
		fileParticles >> particles[i].position.x >> particles[i].position.y >> particles[i].position.z;
	}

	fileParticles.close();
	return particles;
}

double Cube(double number)
{
	return number * number * number;
}

Vector Sum(Vector* sequence, int size)
{
	Vector sum;
	sum.x = .0;
	sum.y = .0;
	sum.z = .0;

	for (int i = 0; i < size; ++i)
	{
		sum = sum + sequence[i];
	}

	return sum;
}

Vector Sum(Vector* sequence, int first, int last)
{
	Vector sum;
	sum.x = .0;
	sum.y = .0;
	sum.z = .0;

	for (int i = first; i < last; ++i)
	{
		sum = sum + sequence[i];
	}

	return sum;
}

